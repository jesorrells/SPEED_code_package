#include "hip/hip_runtime.h"
//Program for performing GPU-accelerated single-photon peak event detection for FLIM
//By Janet Sorrells and Rishee Iyer, 2021
//Contact: janetes2@illinois.edu

/*
Description of variables from Config.txt file for you to provide:
numX - number of pixels along the fast axis  (points per line)
numY - number of pixels along the slow axis  (lines per frame)
numT - number of timepoints sampled per pixel per pulse
numTtot - number of timepoints sampled per pixel per frame
	(ex: if for each pixel you sample at 3.2 GHz, using laser with rep rate 80 MHz, 
	and have a pixel dwell time of 5 us = 400 laser pulses, numTtot = 16000, numT = 40)
nChunks - number of partitions within the image, this is useful for real-time processing so 
	we can transfer data in smaller "chunks" to the PC. You can set it to 1 for post-processing, 
	generally for a 512x512x16000 12-bit dataset, we use nChunks = 32. 
numF - number of frames
Thresh - Intensity threshold (in # of photons) to determine the minimum intensity for lifetime calculations
FileBase - name of the file to open
OutputExt - string to add as an extension to the output data file names

*/

#include "DLLFunctions.cuh"

int main(int argc, char* argv[])
{
	if (argc > 1)
	{
		FILE* ConfigFile = fopen(argv[1], "rt");

		int32_t numX, numY, numT, numTtot, nChunks, numF, numYPerChunk;

		float Thresh;
		char FileBase[1024], OutputExt[1024];

		fscanf(ConfigFile, "%s", &FileBase[0]);
		fscanf(ConfigFile, "%d", &numX);
		fscanf(ConfigFile, "%d", &numY);
		fscanf(ConfigFile, "%d", &numT);
		fscanf(ConfigFile, "%d", &numTtot);
		fscanf(ConfigFile, "%d", &nChunks);
		fscanf(ConfigFile, "%d", &numF);
		fscanf(ConfigFile, "%f", &Thresh);
		fscanf(ConfigFile, "%s", &OutputExt[0]);

		fclose(ConfigFile);

		printf("----------------------------------------------------\n    FAST FLIM POST PROCESSING LOG     \n----------------------------------------------------\n");

		char DataFileName[1024], OutputMPMFile[1024], OutputFLIMFile[1024], OutputGFile[1024], OutputSFile[1024], OutputLogFile[1024];
		strcpy(DataFileName, FileBase);
		strcat(DataFileName, "_FLIMRaw.bin");

		strcpy(OutputMPMFile, FileBase);
		strcat(OutputMPMFile, OutputExt);
		strcat(OutputMPMFile, "_MPMOut.bin");

		strcpy(OutputFLIMFile, FileBase);
		strcat(OutputFLIMFile, OutputExt);
		strcat(OutputFLIMFile, "_FLIMOut.bin");

		strcpy(OutputGFile, FileBase);
		strcat(OutputGFile, OutputExt);
		strcat(OutputGFile, "_GOut.bin");

		strcpy(OutputSFile, FileBase);
		strcat(OutputSFile, OutputExt);
		strcat(OutputSFile, "_SOut.bin");

		strcpy(OutputLogFile, FileBase);
		strcat(OutputLogFile, OutputExt);
		strcat(OutputLogFile, "_LogFile.txt");

		FILE* FIDLog = fopen(OutputLogFile, "wt");

		numYPerChunk = numY / nChunks;

		printf("Raw data file:    %s\n", DataFileName);
		printf("Output MPM file:  %s\n", OutputMPMFile);
		printf("Output FLIM file: %s\n", OutputFLIMFile);
		printf("Output G values:  %s\n", OutputGFile);
		printf("Output S values:  %s\n", OutputSFile);
		printf("Output Log:       %s\n", OutputLogFile);
		printf("%d x %d x %d x %d\n", numTtot, numX, numY, numF);
		printf("%d points per pulse | %d chunks per frame, %d lines per chunk\n", numT, nChunks, numYPerChunk);
		printf("Threshold: %1.2f\n", Thresh);

		fprintf(FIDLog, "Raw data file:    %s\n", DataFileName);
		fprintf(FIDLog, "Output MPM file:  %s\n", OutputMPMFile);
		fprintf(FIDLog, "Output FLIM file: %s\n", OutputFLIMFile);
		fprintf(FIDLog, "Output G values:  %s\n", OutputGFile);
		fprintf(FIDLog, "Output S values:  %s\n", OutputSFile);
		fprintf(FIDLog, "%d x %d x %d x %d\n", numTtot, numX, numY, numF);
		fprintf(FIDLog, "%d points per pulse | Average of %d frames\n", numT, nChunks);
		fprintf(FIDLog, "Threshold: %1.2f\n", Thresh);

		//Initialize and start GPU
		hipError_t Ret0 = FastFLIMGPU_StartGPU();
		hipError_t Ret1 = FastFLIMGPU_InitializeAndAllocateMemory(&numX, &numY, &numTtot, &numT, &nChunks);
		printf("Initialized (%s)\n", hipGetErrorString(Ret1));


		//Allocate memory
		uint16_t* Input = (uint16_t*)malloc(sizeof(uint16_t) * numX * numY * numTtot);
		float* MPMImage_NoAv = (float*)malloc(sizeof(float) * numX * numY);
		float* OutputMPM = (float*)malloc(sizeof(float) * numX * numY);
		float* OutputFLIM = (float*)malloc(sizeof(float) * numX * numY);
		uint16_t* OutputS = (uint16_t*)malloc(sizeof(uint16_t) * numX * numY);
		uint16_t* OutputG = (uint16_t*)malloc(sizeof(uint16_t) * numX * numY);
		AvgDataType* OutputHistogram = (AvgDataType*)malloc(sizeof(AvgDataType) * numX * numY * numT);

		//Initialize file index and chunk index to 0
		int32_t fidx = 0, chidx = 0;;

		//Open output files
		FILE* FIDData = fopen(DataFileName, "rb");
		FILE* FIDMPM = fopen(OutputMPMFile, "wb");
		FILE* FIDFLIM = fopen(OutputFLIMFile, "wb");
		FILE* FIDG = fopen(OutputGFile, "wb");
		FILE* FIDS = fopen(OutputSFile, "wb");

		clock_t t = clock(), FTrack = 0, FBeg, FEnd, FOverall;
		for (fidx = 0; fidx < numF; fidx++) //loop through frames
		{
			for (chidx = 0; chidx < nChunks; chidx++) //loop through chunks
			{
				fread(Input, sizeof(uint16_t), numX * numYPerChunk * numTtot, FIDData); //read each chunk
				FBeg = clock();
				//Perform peak detection on raw data and average from numX x numYPerChunk x numTtot to numX x numYPerChunk x numT
				hipError_t Ret2 = FastFLIMGPU_ReportFrame(Input, numX, numYPerChunk, numTtot, numT, nChunks, chidx);
				FEnd = clock() - FBeg;
				FTrack += FEnd;
			}
			
			//Once all chunks of a single frame are ready, process the whole frame

			FBeg = clock();

			//Perform shifting, spatial binning, and lifetime calculations on whole frame
			hipError_t Ret2 = FastFLIMGPU_DoFastFLIM(OutputMPM, OutputHistogram, OutputFLIM, OutputG, OutputS,
				numX, numY, numT, &Thresh, 0, "_", 0, "_");

			FEnd = clock() - FBeg;
			FTrack += FEnd;

			//Write output files
			fwrite(OutputFLIM, sizeof(float), numX * numY, FIDFLIM); //mean fluorescence lifetime image, numX x numY
			fwrite(OutputMPM, sizeof(float), numX * numY, FIDMPM); //intensity, numX x numY
			fwrite(OutputHistogram, sizeof(AvgDataType), numX * numY * numT, FIDMPM);
			//OutputHistogram shows the decay for each pixel, numX x numY x numT
			fwrite(OutputG, sizeof(uint16_t), numX * numY, FIDG); //g from phasor analysis, numX x numY
			fwrite(OutputS, sizeof(uint16_t), numX * numY, FIDS); //s from phasor analysis, numX x numY
			
			FOverall = clock() - t;
			printf("\rFinished %d/%d frames in %1.3f seconds...                        ", fidx + 1, numF, ((float)FOverall) / CLOCKS_PER_SEC);
		}

		fprintf(FIDLog, "\nJust processing took %1.3f seconds.\n", ((float)FTrack) / CLOCKS_PER_SEC);
		fprintf(FIDLog, "The whole shebang took %1.3f seconds.\n", ((float)FOverall) / CLOCKS_PER_SEC);

		printf("\nJust processing took %1.3f seconds...\n", ((float)FTrack) / CLOCKS_PER_SEC);

		hipError_t Ret3 = FastFLIMGPU_DestroyEverything();
		printf("\nDestroyed everything (%s)\n", hipGetErrorString(Ret3));
		fclose(FIDData);
		fclose(FIDFLIM);
		fclose(FIDMPM);
		fclose(FIDS);
		fclose(FIDG);

		free(Input);
		free(MPMImage_NoAv);
		free(OutputMPM);
		free(OutputFLIM);
		free(OutputG);
		free(OutputS);
		free(OutputHistogram);
	}
	else
	{
	printf("YOU DIDN'T SEND ANY ARGUMENTS!");
	}
	return 0;
}